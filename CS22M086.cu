#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
using namespace std;


ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/

__global__ void kernel2(int MIN,int level,int V,int L,int *d_max,int *d_offset,int *d_csrList,int *d_apr,int* d_aid,int* d_activeVertex){
    __shared__ int count;//local count of a block
    __shared__ int s_max;//local max of a block

    int MAX;
    
    MAX = d_max[blockIdx.x];
    int thid = MIN + blockIdx.x*blockDim.x + threadIdx.x;
    
    int start,end;
    
    int th_max;

    th_max = -1;

    s_max = -1;

    __syncthreads();

    //condition to check valid thid
    if(thid<=MAX && thid<V){
        start = d_offset[thid];//start index of csrlist
        end = d_offset[thid+1];//end index of csrlist

        //condition ot heck if a thread is active. Active Threads increase aid of their adjacent nodes by 1.
        if(d_aid[thid]>=d_apr[thid]){
            if(thid==MIN || thid==MAX ||(thid!=MIN && thid!=MAX && (d_aid[thid-1]>=d_apr[thid-1] || d_aid[thid+1]>=d_apr[thid+1]) ) ){
                atomicAdd(&count,1);//add 1 to shared count of the block
                for(int i=start;i<end;i++){
                    atomicAdd(&d_aid[d_csrList[i]],1);
                }
            }
        }
        //All the  thread of a level compute locally the max vertex id they are adjacent to in next level.
        for(int i=start;i<end;i++){
            int temp1;
            temp1 = d_csrList[i];
            //if(th_min>temp1) th_min = temp1;
            if(th_max<temp1) th_max = temp1;
        }

        //All the threads of block compute max vertex adjacnet in next level by threads in a block.
        atomicMax(&s_max,th_max);
        
        //By now s_max contains max vertex adjacent in next level by threads in a block and active threads have increased adjacent vertex aid by 1. 
        // Count contains number of active vertices in a block for current particular level.
        __syncthreads();
        if(threadIdx.x==0){
            
            //Store max vertex of a block to its corresponding position in d_max.
            d_max[blockIdx.x]=s_max;

            //Add count to d_activeVertex of the current level.
            atomicAdd(&d_activeVertex[level],count);
            count=0;
        }
    }

}

    
/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    // int *d_activeVertex;
	// hipMalloc(&d_activeVertex, L*sizeof(int));


/***Important***/

// Initialize d_aid array to zero for each vertex
// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/

int *d_activeVertex;// Store number of active vertex for all level.
hipMalloc(&d_activeVertex, L*sizeof(int));
//Initialize number of active vertex for all levels as 0.
hipMemset(d_activeVertex,0,L*sizeof(int));
//Initialize aid for all vertices as 0.
hipMemset(d_aid,0,V*sizeof(int));


//d_max to store max reachable vertex of all the vertices in a block for a level.
//h_max cpu copy of d_max to compute max reachble vertex of next level.
//temp_max to intilize d_max before each krnel call for a level.
int *d_max,*h_max,*temp_max;

//min to store first vertex of a level.
//max to store last vertex of a level.
int min,max;
min=0;
max=-1;
//Compute last vertex of level 1.
for(int i=0;i<V;i++){
    if(h_apr[i]!=0) break;
    max++;
}
int num_block,vertex;
//calculate kerenel lauch parameters.
vertex = max+1;
num_block = ceil((float)vertex/1024);
h_max = (int *)malloc(10 * sizeof(int));
temp_max = (int *)malloc(10 * sizeof(int));

memset(h_max,-1,10*sizeof(int));

hipMalloc(&d_max,10*sizeof(int));

//hipMemcpy(d_max,&max,sizeof(int),hipMemcpyHostToDevice);

dim3 blocksize(1024,1,1);

//temp variable to store last index of level for next iteration.
int t_max=max;
for(int i=0;i<L;i++){
    
    t_max=-1;
    //caculate lauch parameters for kernle launch.
    num_block = ceil((float)vertex/1024);
    
    //initialize temp_max
    memset(temp_max,-1,10*sizeof(int));
    
    //intialize d_max
    for(int j=0;j<num_block;j++){
        temp_max[j]=max;    
    }
    hipMemcpy(d_max,temp_max,10 * sizeof(int),hipMemcpyHostToDevice);
    
    
    dim3 gridsize(num_block,1,1);
    // kernel lauch of a level.
    kernel2<<<gridsize,blocksize>>>(min,i, V, L, d_max, d_offset, d_csrList, d_apr, d_aid, d_activeVertex);
    
    //Copy d_max to h_max and calcuate last vertex of next level. 
    hipMemcpy(h_max,d_max,10 * sizeof(int),hipMemcpyDeviceToHost);
    for(int j=0;j<num_block;j++){
        
        if(h_max[j]>t_max) t_max = h_max[j];
    }
    //update first and last index of next level.
    min = max+1;
    max = t_max;
    //reset h_max 
    memset(h_max,-1,10*sizeof(int));
    
    //calculate threads needed for next level.
    vertex = max-min+1;
}


/********************************END OF CODE AREA**********************************/
double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host

hipMemcpy(h_activeVertex,d_activeVertex,L*sizeof(int),hipMemcpyDeviceToHost);

char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}
